#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/compress_inner_product_layer.hpp"

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include <cmath>

namespace caffe {

// The constant NUM_THREADS should be equal to the value in CCMomentCalc
template <typename Dtype>
__global__ void CCMomentCollect(const int n, const Dtype* wb, const Dtype* mask,
    Dtype* mu, Dtype* std, unsigned int* count ) {  
  const int NUM_THREADS = 512;  
  __shared__ Dtype param [4*NUM_THREADS]; 
  __shared__ unsigned int tcount [2*NUM_THREADS];   
  unsigned int t = threadIdx.x; 
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  if (s+t < n){
    param[t] = fabs(mask[s+t]*wb[s+t]);
    param[t+2*NUM_THREADS] = mask[s+t]*wb[s+t]*wb[s+t];
    if(mask[s+t]*wb[s+t]!=0) tcount[t] = 1;
    else tcount[t] = 0;
  }
  else{
    param[t] = 0;param[t+2*NUM_THREADS] = 0;tcount[t] = 0;
  }
  if (s+t+NUM_THREADS < n){
    param[t+NUM_THREADS] = fabs(mask[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS]);
    param[t+3*NUM_THREADS] = mask[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS];
    if(mask[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS]!=0) tcount[t+NUM_THREADS] = 1;
    else tcount[t+NUM_THREADS] = 0;
  }
  else{
    param[t+NUM_THREADS] = 0;param[t+3*NUM_THREADS] = 0;tcount[t+NUM_THREADS] = 0;  
  }
  __syncthreads(); 
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      param[t] += param[t+stride]; 
      param[t+2*NUM_THREADS] += param[t+2*NUM_THREADS+stride];
      tcount[t] += tcount[t+stride];
    }
    __syncthreads();  
  }
  if (t == 0){
    mu   [blockIdx.x] = param[0];
    std  [blockIdx.x] = param[2*NUM_THREADS];
    count[blockIdx.x] = tcount[0]; 
  }      
}

// The constant NUM_THREADS should be equal to the value in CCMomentCalc
template <typename Dtype>
__global__ void CCNzeroCollect(const int n, const Dtype* mask, unsigned int* count ) {  
  const int NUM_THREADS = 512;  
  __shared__ unsigned int tcount [2*NUM_THREADS];   
  unsigned int t = threadIdx.x; 
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  tcount[t] = 0;
  if (s+t < n && mask[s+t]!=0){
    tcount[t] = 1;
  }
  tcount[t+NUM_THREADS] = 0;
  if (s+t+NUM_THREADS < n && mask[s+t+NUM_THREADS]!=0){
    tcount[t+NUM_THREADS] = 1;
  }
  __syncthreads(); 
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      tcount[t] += tcount[t+stride];
    }
    __syncthreads();  
  }
  if (t == 0){
    count[blockIdx.x] = tcount[0]; 
  }     
}

template <typename Dtype>
__global__ void CCMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype mu, Dtype std, Dtype r) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index]==1 && fabs(wb[index])<=0.9*max(mu+r*std,Dtype(0))) 
      mask[index] = 0;
    else if (mask[index]==0 && fabs(wb[index])>1.1*max(mu+r*std,Dtype(0)))
      mask[index] = 1;
  }
}

template <typename Dtype>
__global__ void CCMaskApply(const int n, const Dtype* wb,
    const Dtype* mask, Dtype* wb_t) {
  CUDA_KERNEL_LOOP(index, n) {
    wb_t[index] = wb[index] * mask[index];    
  }
}

template <typename Dtype>
void CCMomentCalc(const int n, const Dtype* wb, const Dtype* mask, Dtype* mu, Dtype* std, unsigned int* ncount){ 
  const unsigned int NUM_THREADS = 512;
  Dtype* pmu_g; Dtype* pstd_g; unsigned int* pncount_g;
  Dtype* pmu_c; Dtype* pstd_c; unsigned int* pncount_c;
  int num_p = (n+(NUM_THREADS<<1)-1)/(NUM_THREADS<<1);  
  hipMalloc(&pmu_g, sizeof(Dtype)  * num_p);
  hipMalloc(&pstd_g, sizeof(Dtype) * num_p);
  hipMalloc(&pncount_g, sizeof(unsigned int) * num_p);
  pmu_c = (Dtype*) malloc(num_p * sizeof(Dtype));
  pstd_c = (Dtype*) malloc(num_p * sizeof(Dtype)); 
  pncount_c = (unsigned int*) malloc(num_p * sizeof(unsigned int));      
  CCMomentCollect<Dtype><<<num_p,NUM_THREADS>>>(n, wb, mask, pmu_g, pstd_g, pncount_g);
  CUDA_POST_KERNEL_CHECK; 
  hipMemcpy(pmu_c, pmu_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pstd_c, pstd_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pncount_c, pncount_g, sizeof(unsigned int) * num_p, hipMemcpyDeviceToHost);      
  for (int i = 0; i < num_p; i++) {
    *mu += pmu_c[i];*std += pstd_c[i];*ncount += pncount_c[i];
  }       
  hipFree(pmu_g);hipFree(pstd_g);hipFree(pncount_g);
  free(pmu_c);free(pstd_c);free(pncount_c);
}

template <typename Dtype>
void CCNZeroCalc(const int n, const Dtype* mask, unsigned int* ncount ){   
  const unsigned int NUM_THREADS = 512;
  unsigned int* pncount_g;
  unsigned int* pncount_c;
  int num_p = (n+(NUM_THREADS<<1)-1)/(NUM_THREADS<<1);  
  hipMalloc(&pncount_g, sizeof(unsigned int) * num_p);
  pncount_c = (unsigned int*) malloc(num_p * sizeof(unsigned int));      
  CCNzeroCollect<Dtype><<<num_p,NUM_THREADS>>>(n, mask, pncount_g);
  CUDA_POST_KERNEL_CHECK; 
  hipMemcpy(pncount_c, pncount_g, sizeof(unsigned int) * num_p, hipMemcpyDeviceToHost);      
  for (int i = 0; i < num_p; i++) {
    *ncount += pncount_c[i];
  }       
  hipFree(pncount_g);
  free(pncount_c);
}

template <typename Dtype>
__global__ void absdata(const int n, Dtype* mask ) {
  CUDA_KERNEL_LOOP (index, n) {
    mask[index] = fabs(mask[index]);
  }
}

template <typename Dtype>
__global__ void CUnewMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype cutLeft, Dtype cutRight) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index]==1 && fabs(wb[index])<= max(cutLeft,Dtype(0))) 
      mask[index] = 0;
    else if (mask[index]==0 && fabs(wb[index])> max(cutRight,Dtype(0)))
      mask[index] = 1;
  }
}

template <typename Dtype>
__global__ void CUmaskCombine(const int n, const Dtype* lastMask, Dtype* mask) {
  CUDA_KERNEL_LOOP(index, n) {
    if (lastMask[index]== 0 ) 
      mask[index] = 0;
  }
}

template <typename Dtype>
__global__ void CCMaskAdjust(const int n, Dtype* mask) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index] > 0.5) {
      mask[index] = 1;
    }
    else {
      mask[index] = 0;
    }
  }
}

template <typename Dtype>
int partition(Dtype * data, int p ,int r) {
  Dtype x = data[r];
  Dtype temp;
  int i = p - 1;
  for(int j = p; j<r; j++) {
    if (data[j] <= x)
    {
      i = i + 1;
      temp = data[i];
      data[i] = data[j];
      data[j] = temp;
    }
  }
  temp = data[i+1];
  data[i+1] = data[r];
  data[r] = temp;
  return i+1;
}

template <typename Dtype>
Dtype findMedian(Dtype * data, int p, int r, int i){
  if (p == r)
  {
    return data[p];
  }
  int q = partition(data, p, r);
  int k = q - p + 1;
  if (i == k)
  {
    return data[q];
  }
  else if (i < k)
  {
    return findMedian(data, p, q - 1, i);
  }
  else
  {
    return findMedian(data, q+1, r, i-k);
  }
}

template <typename Dtype>
void CInnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {    

  const Dtype* weight = this->blobs_[0]->mutable_gpu_data();  
  Dtype* weightMask = this->blobs_[2]->mutable_gpu_data();
  Dtype* weightTmp = this->weight_tmp_.mutable_gpu_data();  
  const Dtype* bias = NULL;
  Dtype* biasMask = NULL;
  Dtype* biasTmp = NULL;
  if (this->bias_term_) {  
    bias = this->blobs_[1]->mutable_gpu_data();   
    biasMask = this->blobs_[3]->mutable_gpu_data();
    biasTmp = this->bias_tmp_.mutable_gpu_data();
  }   

  CCMaskAdjust<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[2]->count()),
    CAFFE_CUDA_NUM_THREADS>>>(this->blobs_[2]->count(), weightMask);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {
    CCMaskAdjust<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
    CAFFE_CUDA_NUM_THREADS>>>(this->blobs_[3]->count(), biasMask);
    CUDA_POST_KERNEL_CHECK;
  }

  if (this->phase_ == TRAIN){
		// Calculate the mean and standard deviation of learnable parameters 		
    if ( this->iter_==0){


        /* code */
      
      this->rand_weight_m_.Reshape(this->blobs_[2]->shape());
      this->rand_bias_m_.Reshape(this->blobs_[3]->shape());
      this->rand_weight_m_.CopyFrom(*(this->blobs_[2]));
      this->rand_bias_m_.CopyFrom(*(this->blobs_[3]));

      // this->crate = 3;
      // if (false)
      // {
      Blob<Dtype> wsort(this->blobs_[0]->shape());
      Blob<Dtype> bsort(this->blobs_[1]->shape());
      wsort.CopyFrom(*(this->blobs_[0]));
      bsort.CopyFrom(*(this->blobs_[1]));

      absdata<Dtype><<<CAFFE_GET_BLOCKS(wsort.count()),CAFFE_CUDA_NUM_THREADS>>>(
         wsort.count(), wsort.mutable_gpu_data());
      CUDA_POST_KERNEL_CHECK;
      absdata<Dtype><<<CAFFE_GET_BLOCKS(bsort.count()),CAFFE_CUDA_NUM_THREADS>>>(
         bsort.count(), bsort.mutable_gpu_data());
      CUDA_POST_KERNEL_CHECK;

      const Dtype* cpuWeightClass = this->blobs_[2]->cpu_data();
      const Dtype* cpuBiasClass = this->blobs_[3]->cpu_data();

      int paramnums = wsort.count() + bsort.count();

      Dtype* allparams = (Dtype*) malloc(paramnums * sizeof(Dtype));

      int temcount = 0;
      for (int i = 0; i < wsort.count(); ++i)
      {
        if (cpuWeightClass[i] > 0)
        {
          allparams[temcount] = wsort.mutable_cpu_data()[i];
          temcount++;
        }
      }
      for (int i = 0; i < bsort.count(); ++i)
      {
        if (cpuBiasClass[i] > 0)
        {
          allparams[temcount] = bsort.mutable_cpu_data()[i];
          temcount++;
        }
      }
      for (int i = temcount; i < paramnums; ++i)
      {
        allparams[i] = 0;
      }

      std::cout<<"temcount:"<<temcount<<std::endl;


      this->cutLeft = findMedian(allparams, 0, temcount - 1, int(temcount * this->crate));
      this->cutRight = findMedian(allparams, 0, temcount - 1, int(temcount * (this->crate + 0.05)));
      free(allparams);

      std::cout<<"cutLeft:"<<this->cutLeft<<std::endl;
      std::cout<<"cutRight:"<<this->cutRight<<std::endl;
      // }

      // unsigned int ncount = 0;
      // CCMomentCalc(this->blobs_[0]->count(), weight, weightMask, &mu, &std, &ncount);
      // if (this->bias_term_) {  
      //   CCMomentCalc(this->blobs_[1]->count(), bias, biasMask, &mu, &std, &ncount); 
      // }     
      // this->mu /= ncount; this->std -= ncount*mu*mu; 
      // this->std /= ncount; this->std = sqrt(std);  
      // LOG(INFO)<<mu<<"  "<<std<<"  "<<ncount<<"\n";     


    }
		
		// Demonstrate the sparsity of compressed fully-connected layer
		/********************************************************/
		if(this->iter_%100==0){
      unsigned int wNoneZero = 0;
      unsigned int wAll = 0;
      unsigned int bNoneZero = 0;
      unsigned int bAll = 0;
			CCNZeroCalc(this->blobs_[0]->count(), weightMask, &wNoneZero);
      wAll = this->blobs_[0]->count();
			if (this->bias_term_) {  
				CCNZeroCalc(this->blobs_[1]->count(), biasMask, &bNoneZero); 
        bAll = this->blobs_[1]->count();  
			}
			//LOG(INFO)<<ncount<<"\n";  			
      LOG(INFO)<<wNoneZero<<"  "<<wAll<<"  "<<bNoneZero<<"  "<<bAll<<"\n"; 
		}	
		/********************************************************/		
		
		// Calculate the weight mask and bias mask with probability
    Dtype r = static_cast<Dtype>(rand())/static_cast<Dtype>(RAND_MAX);
    if (pow(1+(this->gamma)*(this->iter_),-(this->power))>r && (this->iter_)<(this->iter_stop_)) { 
      // CCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
      //   CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, this->mu, this->std, this->crate);
      // CUDA_POST_KERNEL_CHECK;    
      CUnewMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, this->cutLeft, this->cutRight);
      CUDA_POST_KERNEL_CHECK;  
      if (this->bias_term_) {  
        // CCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
        //   CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, this->mu, this->std, this->crate);
        // CUDA_POST_KERNEL_CHECK; 
        CUnewMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
          CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, this->cutLeft, this->cutRight);
        CUDA_POST_KERNEL_CHECK;  
      }    
    }

    CUmaskCombine<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), this->rand_weight_m_.gpu_data(),
      weightMask);
    CUDA_POST_KERNEL_CHECK;
    CUmaskCombine<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), this->rand_bias_m_.gpu_data(),
      biasMask);
    CUDA_POST_KERNEL_CHECK;   
  }  
  
  // Calculate the current (masked) weight and bias
  CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
    CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, weightTmp);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {  
    CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, biasTmp);
    CUDA_POST_KERNEL_CHECK;  
  } 
   
	// Forward calculation with (masked) weight and bias 
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weightTmp, bottom_data, (Dtype)0., top_data);
    if (this->bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            biasTmp, top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
                          bottom_data, weightTmp, (Dtype)0., top_data);
    if (this->bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            biasTmp, (Dtype)1., top_data);
  }

  this->iter_++;
}

template <typename Dtype>
void CInnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  if (this->param_propagate_down_[0]) {
		const Dtype* weightMask = this->blobs_[2]->gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
		CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[2]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[2]->count(), weight_diff, weightMask, weight_diff);
    CUDA_POST_KERNEL_CHECK; 
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
        top_diff, bottom_data, (Dtype)1., weight_diff);
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
		const Dtype* biasMask = this->blobs_[3]->gpu_data();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    // Gradient with respect to bias
    CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[3]->count(), bias_diff, biasMask, bias_diff);
    CUDA_POST_KERNEL_CHECK; 		
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,bias_diff);
  }	
  if (propagate_down[0]) {
		const Dtype* weightTmp = this->weight_tmp_.gpu_data();        
    // Gradient with respect to bottom data
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
        top_diff, weightTmp, (Dtype)0.,
        bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CInnerProductLayer);

}  // namespace caffe
