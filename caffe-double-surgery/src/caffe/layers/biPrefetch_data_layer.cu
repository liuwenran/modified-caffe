#include <vector>

#include "caffe/layers/double_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BiPrefetchDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  DoubleBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  top[1]->ReshapeLike(batch->secondData_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  caffe_copy(batch->secondData_.count(), batch->secondData_.gpu_data(),
    top[1]->mutable_gpu_data());

  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[2]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[2]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BiPrefetchDataLayer);

}  // namespace caffe
