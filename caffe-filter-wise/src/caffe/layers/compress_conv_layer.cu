#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/compress_conv_layer.hpp"

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include <cmath>

namespace caffe {

// The constant NUM_THREADS should be equal to the value in CCMomentCalc
template <typename Dtype>
__global__ void CCMomentCollect(const int n, const Dtype* wb, const Dtype* mask,
    Dtype* mu, Dtype* std, unsigned int* count ) {  
  const int NUM_THREADS = 512;  
  __shared__ Dtype param [4*NUM_THREADS]; 
  __shared__ unsigned int tcount [2*NUM_THREADS];   
  unsigned int t = threadIdx.x; 
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  if (s+t < n){
    param[t] = fabs(mask[s+t]*wb[s+t]);
    param[t+2*NUM_THREADS] = mask[s+t]*wb[s+t]*wb[s+t];
    if(mask[s+t]*wb[s+t]!=0) tcount[t] = 1;
    else tcount[t] = 0;
  }
  else{
    param[t] = 0;param[t+2*NUM_THREADS] = 0;tcount[t] = 0;
  }
  if (s+t+NUM_THREADS < n){
    param[t+NUM_THREADS] = fabs(mask[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS]);
    param[t+3*NUM_THREADS] = mask[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS];
    if(mask[s+t+NUM_THREADS]*wb[s+t+NUM_THREADS]!=0) tcount[t+NUM_THREADS] = 1;
    else tcount[t+NUM_THREADS] = 0;
  }
  else{
    param[t+NUM_THREADS] = 0;param[t+3*NUM_THREADS] = 0;tcount[t+NUM_THREADS] = 0;  
  }
  __syncthreads(); 
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      param[t] += param[t+stride]; 
      param[t+2*NUM_THREADS] += param[t+2*NUM_THREADS+stride];
      tcount[t] += tcount[t+stride];
    }
    __syncthreads();  
  }
  if (t == 0){
    mu   [blockIdx.x] = param[0];
    std  [blockIdx.x] = param[2*NUM_THREADS];
    count[blockIdx.x] = tcount[0]; 
  }      
}

// The constant NUM_THREADS should be equal to the value in CCMomentCalc
template <typename Dtype>
__global__ void CCNzeroCollect(const int n, const Dtype* mask, unsigned int* count ) {  
  const int NUM_THREADS = 512;  
  __shared__ unsigned int tcount [2*NUM_THREADS];   
  unsigned int t = threadIdx.x; 
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  tcount[t] = 0;
  if (s+t < n && mask[s+t]!=0){
    tcount[t] = 1;
  }
  tcount[t+NUM_THREADS] = 0;
  if (s+t+NUM_THREADS < n && mask[s+t+NUM_THREADS]!=0){
    tcount[t+NUM_THREADS] = 1;
  }
  __syncthreads(); 
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      tcount[t] += tcount[t+stride];
    }
    __syncthreads();  
  }
  if (t == 0){
    count[blockIdx.x] = tcount[0]; 
  }     
}

template <typename Dtype>
__global__ void CCMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype mu, Dtype std, Dtype r) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index]==1 && fabs(wb[index])<=0.9*max(mu+r*std,Dtype(0))) 
      mask[index] = 0;
    else if (mask[index]==0 && fabs(wb[index])>1.1*max(mu+r*std,Dtype(0)))
      mask[index] = 1;
  }
}

template <typename Dtype>
__global__ void CCMaskApply(const int n, const Dtype* wb,
    const Dtype* mask, Dtype* wb_t) {
  CUDA_KERNEL_LOOP(index, n) {
    wb_t[index] = wb[index] * mask[index];    
  }
}

template <typename Dtype>
void CCMomentCalc(const int n, const Dtype* wb, const Dtype* mask, Dtype* mu, Dtype* std, unsigned int* ncount){ 
  const unsigned int NUM_THREADS = 512;
  Dtype* pmu_g; Dtype* pstd_g; unsigned int* pncount_g;
  Dtype* pmu_c; Dtype* pstd_c; unsigned int* pncount_c;
  int num_p = (n+(NUM_THREADS<<1)-1)/(NUM_THREADS<<1);  
  hipMalloc(&pmu_g, sizeof(Dtype)  * num_p);
  hipMalloc(&pstd_g, sizeof(Dtype) * num_p);
  hipMalloc(&pncount_g, sizeof(unsigned int) * num_p);
  pmu_c = (Dtype*) malloc(num_p * sizeof(Dtype));
  pstd_c = (Dtype*) malloc(num_p * sizeof(Dtype)); 
  pncount_c = (unsigned int*) malloc(num_p * sizeof(unsigned int));      
  CCMomentCollect<Dtype><<<num_p,NUM_THREADS>>>(n, wb, mask, pmu_g, pstd_g, pncount_g);
  CUDA_POST_KERNEL_CHECK; 
  hipMemcpy(pmu_c, pmu_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pstd_c, pstd_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pncount_c, pncount_g, sizeof(unsigned int) * num_p, hipMemcpyDeviceToHost);      
  for (int i = 0; i < num_p; i++) {
    *mu += pmu_c[i];*std += pstd_c[i];*ncount += pncount_c[i];
  }       
  hipFree(pmu_g);hipFree(pstd_g);hipFree(pncount_g);
  free(pmu_c);free(pstd_c);free(pncount_c);
}

template <typename Dtype>
void CCNZeroCalc(const int n, const Dtype* mask, unsigned int* ncount ){  
  const unsigned int NUM_THREADS = 512;
  unsigned int* pncount_g;
  unsigned int* pncount_c;
  int num_p = (n+(NUM_THREADS<<1)-1)/(NUM_THREADS<<1);  
  hipMalloc(&pncount_g, sizeof(unsigned int) * num_p);
  pncount_c = (unsigned int*) malloc(num_p * sizeof(unsigned int));      
  CCNzeroCollect<Dtype><<<num_p,NUM_THREADS>>>(n, mask, pncount_g);
  CUDA_POST_KERNEL_CHECK; 
  hipMemcpy(pncount_c, pncount_g, sizeof(unsigned int) * num_p, hipMemcpyDeviceToHost);      
  for (int i = 0; i < num_p; i++) {
    *ncount += pncount_c[i];
  }       
  hipFree(pncount_g);
  free(pncount_c);
}

template <typename Dtype>
__global__ void absdata(const int n, Dtype* mask ) {
  CUDA_KERNEL_LOOP (index, n) {
    mask[index] = fabs(mask[index]);
  }
}

template <typename Dtype>
__global__ void CUnewMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype cutLeft, Dtype cutRight) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index]==1 && fabs(wb[index])<= max(cutLeft,Dtype(0))) 
      mask[index] = 0;
    else if (mask[index]==0 && fabs(wb[index])> max(cutRight,Dtype(0)))
      mask[index] = 1;
  }
}


template <typename Dtype>
__global__ void CUmaskCombine(const int n, const Dtype* lastMask, Dtype* mask) {
  CUDA_KERNEL_LOOP(index, n) {
    if (lastMask[index]== 0 ) 
      mask[index] = 0;
  }
}

template <typename Dtype>
__global__ void CCMaskAdjust(const int n, Dtype* mask) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index] > 0.5) {
      mask[index] = 1;
    }
    else {
      mask[index] = 0;
    }
  }
}

template <typename Dtype>
int partition(Dtype * data, int p ,int r) {
  Dtype x = data[r];
  Dtype temp;
  int i = p - 1;
  for(int j = p; j<r; j++) {
    if (data[j] <= x)
    {
      i = i + 1;
      temp = data[i];
      data[i] = data[j];
      data[j] = temp;
    }
  }
  temp = data[i+1];
  data[i+1] = data[r];
  data[r] = temp;
  return i+1;
}

template <typename Dtype>
Dtype findMedian(Dtype * data, int p, int r, int i){
  if (p == r)
  {
    return data[p];
  }
  int q = partition(data, p, r);
  int k = q - p + 1;
  if (i == k)
  {
    return data[q];
  }
  else if (i < k)
  {
    return findMedian(data, p, q - 1, i);
  }
  else
  {
    return findMedian(data, q+1, r, i-k);
  }
}


template <typename Dtype>
void CConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  // const Dtype* weight = this->blobs_[0]->mutable_gpu_data();  
  Dtype* weightMask = this->blobs_[2]->mutable_gpu_data();
  Dtype* weightTmp = this->weight_tmp_.mutable_gpu_data(); 
  // const Dtype* bias = NULL;
  Dtype* biasMask = NULL;
  Dtype* biasTmp = NULL;   
  if (this->bias_term_) {  
    // bias = this->blobs_[1]->mutable_gpu_data();   
    biasMask = this->blobs_[3]->mutable_gpu_data();
    biasTmp = this->bias_tmp_.mutable_gpu_data();
  }


  CCMaskAdjust<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[2]->count()),
    CAFFE_CUDA_NUM_THREADS>>>(this->blobs_[2]->count(), weightMask);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {
    CCMaskAdjust<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
    CAFFE_CUDA_NUM_THREADS>>>(this->blobs_[3]->count(), biasMask);
    CUDA_POST_KERNEL_CHECK;
  }

  std::vector<int> weight_shape = this->blobs_[2]->shape();
  int filter_size = 1;
  for (int i = 1; i < weight_shape.size(); ++i)
  {
    filter_size = filter_size * weight_shape[i];
  }
  
  if (this->phase_ == TRAIN){
    // Calculate the mean and standard deviation of learnable parameters 

    if(this->iter_%100==0){
      unsigned int wNoneZero = 0;
      unsigned int wAll = 0;
      unsigned int bNoneZero = 0;
      unsigned int bAll = 0;
      CCNZeroCalc(this->blobs_[0]->count(), this->blobs_[2]->mutable_gpu_data(), &wNoneZero);
      wAll = this->blobs_[0]->count();
      if (this->bias_term_) {  
        CCNZeroCalc(this->blobs_[1]->count(), this->blobs_[3]->mutable_gpu_data(), &bNoneZero); 
        bAll = this->blobs_[1]->count();  
      }
      //LOG(INFO)<<ncount<<"\n";        
      LOG(INFO)<<wNoneZero<<"  "<<wAll<<"  "<<bNoneZero<<"  "<<bAll<<"\n"; 

    } 

    if ( this->iter_==0){

      std::vector<int> statistics;
      std::vector<int> notzero;
      statistics.resize(weight_shape[0]);
      
      Dtype * forfiltermask = this->blobs_[2]->mutable_cpu_data();
      Dtype * forbiasmask = this->blobs_[3]->mutable_cpu_data();
      Dtype * forfilterWeight = this->blobs_[0]->mutable_cpu_data();
      Dtype * forbiasWeight = this->blobs_[1]->mutable_cpu_data();

      LOG(INFO)<<"filter_size: "<<filter_size<<"\n"; 

      for (int i = 0; i < weight_shape[0]; ++i)
      {
        int offset = this->blobs_[2]->offset(i);
        // std::cout<<"offset "<<offset<<std::endl;
        Dtype * filter_start = forfiltermask + offset;
        // int temp = 0;
        // std::cout<<"cin come:"<<std::endl;
        // std::cin>>temp;
        int count = 0;
        // std::cout<<"filter_size "<<filter_size<<std::endl;
        for (int j = 0; j < filter_size; ++j)
        {
          // std::cout<<filter_start[j]<<" ";
          if(filter_start[j] > 0)
          {
            count++;
          }
        }
        // std::cout<<std::endl<<"count "<<count<<std::endl;
        statistics[i] = count;
        if(count > 0)
        {
          notzero.push_back(count);
        }
      }

      sort(notzero.begin(), notzero.end());


      int lowerInd = notzero.size() * this->abandonpercent;
      if (lowerInd < 1)
      {
        lowerInd = 1;
      }
      int lowerBound = notzero[lowerInd - 1];

      std::cout<<"notzero size: "<<notzero.size()<<" lowerInd: "<<lowerInd<<" lowerBound: "<<lowerBound<<std::endl;

      int abandon_num = 0;
      for (int i = 0; i < weight_shape[0]; ++i)
      {
        int offset = this->blobs_[2]->offset(i);
        Dtype * filter_start = forfiltermask + offset;
        Dtype * filter_weight_start = forfilterWeight + offset;
        if( statistics[i] < lowerBound && statistics[i] < filter_size / 2)
        {
          for (int j = 0; j < filter_size; ++j)
          {
            filter_start[j] = 0;
          }
          forbiasmask[i] = 0;
          abandon_num++;
        }
        else
        {
          for (int j = 0; j < filter_size; ++j)
          {
            if(filter_start[j] < 0.5)
            {
              filter_start[j] = 1;
              filter_weight_start[j] = 0;
            }
          }
          if(forbiasmask[i] < 0.5)
          {
            forbiasmask[i] = 1;
            forbiasWeight[i] = 0;
          }
        }
      }     

      std::cout<<"abandon_num: "<<abandon_num<<std::endl;

      // 
    }

  }   

  CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
    CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), this->blobs_[0]->mutable_gpu_data(),
     this->blobs_[2]->mutable_gpu_data(), weightTmp);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {  
    CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), this->blobs_[1]->mutable_gpu_data(),
       this->blobs_[3]->mutable_gpu_data(), biasTmp);
    CUDA_POST_KERNEL_CHECK;  
  }

  // Forward calculation with (masked) weight and bias 
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + bottom[i]->offset(n), weightTmp,
          top_data + top[i]->offset(n));
      if (this->bias_term_) {
        this->forward_gpu_bias(top_data + top[i]->offset(n), biasTmp);
      }
    }
  }

  this->iter_++;
}


template <typename Dtype>
void CConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weightTmp = this->weight_tmp_.gpu_data();    
  const Dtype* weightMask = this->blobs_[2]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();   
  for (int i = 0; i < top.size(); ++i) {    
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      const Dtype* biasMask = this->blobs_[3]->gpu_data();
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();     
      CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[3]->count(), bias_diff, biasMask, bias_diff);
      CUDA_POST_KERNEL_CHECK;  
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + top[i]->offset(n));
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[2]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[2]->count(), weight_diff, weightMask, weight_diff);
      CUDA_POST_KERNEL_CHECK;       
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + bottom[i]->offset(n),
              top_diff + top[i]->offset(n), weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + top[i]->offset(n), weightTmp,
              bottom_diff + bottom[i]->offset(n));
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CConvolutionLayer);

}  // namespace caffe
